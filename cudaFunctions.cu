#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

// For the GPU to recognize this function
__device__ double func(int i)
{
    double value;
    double result = 0;

    for (int j = 1; j < HEAVY; j++)
    {
        value = (i + 1) * (j % 10);
        result += cos(value);
    }
    return cos(result);
}

// each thread has arr,numElements his own.
__global__ void fFunction(int *arr, int numElements, int *counter)
{

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
        if (func(arr[i]) > 0) // we will update the arr when we have func(arr[i])>0 so we will
            atomicAdd(&counter[0], 1);
    __syncthreads();
}

int computeOnGPU(int *data, int numElements, int *counter)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(int);
    size_t sizeCounter = sizeof(int);
    // Allocate memory on GPU to copy the data from the host
    int *d_A;
    int *d_counter;

    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_counter, sizeCounter);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_A, data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_counter, counter, sizeCounter, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    fFunction<<<blocksPerGrid, threadsPerBlock>>>(d_A, numElements, d_counter);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from GPU to the HOST memory
    err = hipMemcpy(counter, d_counter, sizeCounter, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_A) != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_counter) != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return EXIT_SUCCESS;
}
